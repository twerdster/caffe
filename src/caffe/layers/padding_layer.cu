#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"


namespace caffe {

template <typename Dtype>
inline __device__ Dtype caffe_gpu_atomic_add(const Dtype val, Dtype* address);

template <>
inline __device__
float caffe_gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

// double atomicAdd implementation taken from:
// http://docs.nvidia.com/cuda/cuda-c-programming-guide/#axzz3PVCpVsEG
template <>
inline __device__
double caffe_gpu_atomic_add(const double val, double* address) {
  unsigned long long int* address_as_ull =  // NOLINT(runtime/int)
      // NOLINT_NEXT_LINE(runtime/int)
      reinterpret_cast<unsigned long long int*>(address);
  unsigned long long int old = *address_as_ull;  // NOLINT(runtime/int)
  unsigned long long int assumed;  // NOLINT(runtime/int)
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
        __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

}  // namespace caffe


namespace caffe {


/*
Efficiency note:
The forward and backward processes are just default implementations
taken from the cpu version. The correct way of doing them would probably
be to run a specific kernel for each part of the padding grid 0-8.
To do this effectively without writing 9 kernels for each one you can use
a templated function of the boxId and call the kernels in a loop
from the cpu.
See here for reference of templated function for switch statements:
http://stackoverflow.com/questions/6179295/if-statement-inside-a-cuda-kernel
*/

template <typename Dtype>
__global__ void RepeatPadForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int padded_height, const int padded_width, 
    const int pad_l, const int pad_r, const int pad_t, const int pad_b,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(padded_index, nthreads) {
    const int pw = padded_index % padded_width;
    const int ph = (padded_index / padded_width) % padded_height;
    const int c = (padded_index / padded_width / padded_height) % channels;
    const int n = padded_index / padded_width / padded_height / channels;
    
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;       

    int boxIdx = ((ph<pad_t)?0:(ph<pad_t+height)?1:2)*3 + ((pw<pad_l)?0:(pw<pad_l+width)?1:2);
    int bottom_index = 0; 
    switch (boxIdx){
      case 0: bottom_index =          (0) * width +          (0); break; //Top Left 
      case 1: bottom_index =          (0) * width + (pw - pad_l); break; //Top Center
      case 2: bottom_index =          (0) * width +  (width - 1); break; //Top Right
      case 3: bottom_index = (ph - pad_t) * width +          (0); break; //Center Left
      case 4: bottom_index = (ph - pad_t) * width + (pw - pad_l); break; //Center Center
      case 5: bottom_index = (ph - pad_t) * width +  (width - 1); break; //Center Right
      case 6: bottom_index = (height - 1) * width +          (0); break; //Bottom Left
      case 7: bottom_index = (height - 1) * width + (pw - pad_l); break; //Bottom Center
      case 8: bottom_index = (height - 1) * width +  (width - 1); break; //Bottom Right
    }
            
    top_data[padded_index] = bottom_slice[bottom_index];
  }
}

template <typename Dtype>
__global__ void ConstantPadForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int padded_height, const int padded_width, 
    const int pad_l, const int pad_r, const int pad_t, const int pad_b, const float pad_value,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(padded_index, nthreads) {
    const int pw = padded_index % padded_width;
    const int ph = (padded_index / padded_width) % padded_height;
    const int c = (padded_index / padded_width / padded_height) % channels;
    const int n = padded_index / padded_width / padded_height / channels;
    
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;       

    int boxIdx = ((ph<pad_t)?0:(ph<pad_t+height)?1:2)*3 + ((pw<pad_l)?0:(pw<pad_l+width)?1:2);
    int bottom_index = 0; 
    switch (boxIdx){
      //case 0: bottom_index =          (0) * width +          (0); break; //Top Left 
      //case 1: bottom_index =          (0) * width + (pw - pad_l); break; //Top Center
      //case 2: bottom_index =          (0) * width +  (width - 1); break; //Top Right
      //case 3: bottom_index = (ph - pad_t) * width +          (0); break; //Center Left
      case 4: bottom_index = (ph - pad_t) * width + (pw - pad_l); 
         top_data[padded_index] = bottom_slice[bottom_index];        break; //Center Center
      //case 5: bottom_index = (ph - pad_t) * width +  (width - 1); break; //Center Right
      //case 6: bottom_index = (height - 1) * width +          (0); break; //Bottom Left
      //case 7: bottom_index = (height - 1) * width + (pw - pad_l); break; //Bottom Center
      //case 8: bottom_index = (height - 1) * width +  (width - 1); break; //Bottom Right
      default: top_data[padded_index] = pad_value;               break; 
    }
  }
}


template <typename Dtype>
void PaddingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();

  switch (this->layer_param_.padding_param().pad_method()) {
  case PaddingParameter_PadMethod_REPEAT:
    // NOLINT_NEXT_LINE(whitespace/operators)
    RepeatPadForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, padded_height_, padded_width_, 
        pad_l_, pad_r_, pad_t_, pad_b_, top_data);
    break;
  case PaddingParameter_PadMethod_CONSTANT:
    // NOLINT_NEXT_LINE(whitespace/operators)
    ConstantPadForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, padded_height_, padded_width_, 
        pad_l_, pad_r_, pad_t_, pad_b_, pad_value_, top_data);
    break;
  default:
    LOG(FATAL) << "Unknown padding method (GPU).";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void RepeatPadBackward(const int nthreads,
    const Dtype* const top_diff, const int num, const int channels,
    const int height, const int width, const int padded_height, const int padded_width, 
    const int pad_l, const int pad_r, const int pad_t, const int pad_b,
    Dtype* const bottom_diff) {

  CUDA_KERNEL_LOOP(padded_index, nthreads) {
    const int pw = padded_index % padded_width;
    const int ph = (padded_index / padded_width) % padded_height;
    const int c = (padded_index / padded_width / padded_height) % channels;
    const int n = padded_index / padded_width / padded_height / channels;
    
    Dtype* const bottom_diff_slice =
        bottom_diff + (n * channels + c) * height * width;       

    int bottom_index = 0;

    int boxIdx = ((ph<pad_t)?0:(ph<pad_t+height)?1:2)*3 + ((pw<pad_l)?0:(pw<pad_l+width)?1:2);
    switch (boxIdx){
      case 0: bottom_index =          (0) * width +          (0); break; //Top Left 
      case 1: bottom_index =          (0) * width + (pw - pad_l); break; //Top Center
      case 2: bottom_index =          (0) * width +  (width - 1); break; //Top Right
      case 3: bottom_index = (ph - pad_t) * width +          (0); break; //Center Left
      case 4: bottom_index = (ph - pad_t) * width + (pw - pad_l); break; //Center Center
      case 5: bottom_index = (ph - pad_t) * width +  (width - 1); break; //Center Right
      case 6: bottom_index = (height - 1) * width +          (0); break; //Bottom Left
      case 7: bottom_index = (height - 1) * width + (pw - pad_l); break; //Bottom Center
      case 8: bottom_index = (height - 1) * width +  (width - 1); break; //Bottom Right
    }
    //bottom_diff_slice[bottom_index] += top_diff[padded_index];  
    caffe_gpu_atomic_add(top_diff[padded_index],bottom_diff_slice + bottom_index);
  }
}

template <typename Dtype>
__global__ void ConstantPadBackward(const int nthreads,
    const Dtype* const top_diff, const int num, const int channels,
    const int height, const int width, const int padded_height, const int padded_width, 
    const int pad_l, const int pad_r, const int pad_t, const int pad_b,
    Dtype* const bottom_diff) {

  CUDA_KERNEL_LOOP(padded_index, nthreads) {
    const int pw = padded_index % padded_width;
    const int ph = (padded_index / padded_width) % padded_height;
    const int c = (padded_index / padded_width / padded_height) % channels;
    const int n = padded_index / padded_width / padded_height / channels;
    
    Dtype* const bottom_diff_slice =
        bottom_diff + (n * channels + c) * height * width;       

    int bottom_index = 0;

    int boxIdx = ((ph<pad_t)?0:(ph<pad_t+height)?1:2)*3 + ((pw<pad_l)?0:(pw<pad_l+width)?1:2);
    switch (boxIdx){
    //case 0: bottom_index =           (0) * width_ +           (0); break; //Top Left 
    //case 1: bottom_index =           (0) * width_ + (pw - pad_l_); break; //Top Center
    //case 2: bottom_index =           (0) * width_ +  (width_ - 1); break; //Top Right
    //case 3: bottom_index = (ph - pad_t_) * width_ +           (0); break; //Center Left
      case 4: bottom_index = (ph - pad_t) * width + (pw - pad_l);           //Center Center
              bottom_diff_slice[bottom_index] += top_diff[padded_index];        break; 
    //case 5: bottom_index = (ph - pad_t_) * width_ +  (width_ - 1); break; //Center Right
    //case 6: bottom_index = (height_ - 1) * width_ +           (0); break; //Bottom Left
    //case 7: bottom_index = (height_ - 1) * width_ + (pw - pad_l_); break; //Bottom Center
    //case 8: bottom_index = (height_ - 1) * width_ +  (width_ - 1); break; //Bottom Right
      default: break; // I.e. derivative is zero and has no influence
    }  
  }
}


template <typename Dtype>
void PaddingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int bottom_count = bottom[0]->count();
  const int top_count = top[0]->count();
  caffe_gpu_set(bottom_count, Dtype(0.), bottom_diff);
 
  switch (this->layer_param_.padding_param().pad_method()) {
  case PaddingParameter_PadMethod_REPEAT:
    // NOLINT_NEXT_LINE(whitespace/operators)
    RepeatPadBackward<Dtype><<<CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS>>>(
        top_count, top_diff, top[0]->num(), channels_,
        height_, width_, padded_height_, padded_width_,
        pad_l_, pad_r_, pad_t_, pad_b_,
        bottom_diff);
    break;
  case PaddingParameter_PadMethod_CONSTANT:
    // NOLINT_NEXT_LINE(whitespace/operators)
    ConstantPadBackward<Dtype><<<CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS>>>(
        top_count, top_diff, top[0]->num(), channels_,
        height_, width_, padded_height_, padded_width_, 
        pad_l_, pad_r_, pad_t_, pad_b_, 
        bottom_diff);
    break;
  case PaddingParameter_PadMethod_WRAP:
    // NOLINT_NEXT_LINE(whitespace/operators)
    LOG(FATAL) << "WRAP not implemented";
    break;
  default:
    LOG(FATAL) << "Unknown padding method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(PaddingLayer);


}  // namespace caffe
